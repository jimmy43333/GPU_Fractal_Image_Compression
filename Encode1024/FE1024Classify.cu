#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/core/cuda.hpp>
#include <vector>

#define  N     1024
#define  N2     512
#define  Db      16
#define  Rb       4
#define  Dnum   504  //N2-Rb

using namespace cv;
using namespace std;

//Run on terminal:
//    nvcc FE1024Classify.cu -o FE1024 `pkg-config --cflags --libs opencv` --expt-relaxed-constexpr
//    nvprof ./FE1024 ../Dataset/Image1024/

Mat readRawfile(const char* filename,int width,int height){
    Mat outputimage;
    //read the raw file
    FILE *fp = NULL;
    char *imagedata = NULL;
    int IMAGE_WIDTH = width;
    int IMAGE_HEIGHT = height;
    int framesize = IMAGE_WIDTH * IMAGE_HEIGHT;
    //Open raw Bayer image.
    fp = fopen(filename, "rb");
    if(!fp){
        cout << "read file failure";
        return outputimage;
    }
    //Memory allocation for bayer image data buffer.
    imagedata = (char*) malloc (sizeof(char) * framesize);
    //Read image data and store in buffer.
    fread(imagedata, sizeof(char), framesize, fp);
    //Create Opencv mat structure for image dimension. For 8 bit bayer, type should be CV_8UC1.
    outputimage.create(IMAGE_HEIGHT, IMAGE_WIDTH, CV_8UC1);
    memcpy(outputimage.data, imagedata, framesize);
    free(imagedata);
    fclose(fp);
    return outputimage;
}

bool InitCUDA()
{
    int count;
    hipGetDeviceCount(&count);
    if(count == 0) {
        cout << "There is no device."<< endl;
        return false;
    }
    int i;
    for(i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if(prop.major >= 1) {
                break;
            }
        }
    }
    if(i == count){
        cout << "There is no device supporting CUDA 1.x." << endl;
        return false;
    }
    hipSetDevice(i);
    return true;
}

__device__ void permutation(const int *h,int *a,int H,int R,int k){
    //x,y is the position in the Mat h.
    //R is the size of array a.
    int i1,j1;
    switch(k){
           case 0: 
             for (i1=0; i1<R; i1++)
               for (j1=0; j1<R; j1++)
                 *(a+i1*R+j1) = *(h+i1*H+j1);
             break;
           case 1:
             for (i1=0; i1<R; i1++)
               for (j1=0; j1<R; j1++)
                 *(a+i1*R+j1)= *(h+(R-1-j1)*H+i1);
             break;
           case 2:
             for (i1=0; i1<R; i1++)
               for (j1=0; j1<R; j1++)
                 *(a+i1*R+j1)= *(h+(R-1-i1)*H+R-1-j1);
             break;
           case 3:
             for (i1=0; i1<R; i1++)
               for (j1=0; j1<R; j1++)
                 *(a+i1*R+j1)= *(h+j1*H+R-1-i1);
             break;
                                 /* Reflect w.r.t. y-axis,  then rotate 
                                    counterclockwise 90, 180, 270 degree(s)
                                 */
           case 4:
             for (i1=0; i1<R; i1++)
               for (j1=0; j1<R; j1++)
                 *(a+i1*R+j1)= *(h+i1*H+R-1-j1);
             break;
           case 5:
              for (i1=0; i1<R; i1++)
                for (j1=0; j1<R; j1++)
                  *(a+i1*R+j1)= *(h+(R-1-j1)*H+R-1-i1);
              break;
           case 6:
             for (i1=0; i1<R; i1++)
               for (j1=0; j1<R; j1++)
                 *(a+i1*R+j1)= *(h+(R-1-i1)*H+j1);
             break;
           case 7:
              for (i1=0; i1<R; i1++)
                for (j1=0; j1<R; j1++)
                  *(a+i1*R+j1)= *(h+(j1)*H+i1);
              break;

           } /* end switch */
}

__device__ int Classify(int* inputBlock,int inputSize){
    const int BlockSize = Rb;
    int permu[BlockSize][BlockSize];
    int a1,a2,a3,a4; //Four subblock of the classify block
    int i,j,k;
    int kout;
    //Permutation
    for(k=0;k<8;k++){
        permutation(inputBlock,&permu[0][0],inputSize,BlockSize,k);
        //Calculate a1,a2,a3,a4
        a1=0;
        a2=0;
        a3=0;
        a4=0;
        for(i=0;i<BlockSize;i++){
          for(j=0;j<BlockSize;j++){
              if(i < BlockSize/2 && j< BlockSize/2){
                  a1 += permu[i][j];
              }else if(i< BlockSize/2 && j >= BlockSize/2){
                  a2 += permu[i][j];
              }else if(i >= BlockSize/2 && j < BlockSize/2){
                  a3 += permu[i][j];
              }else{
                  a4 += permu[i][j];
              }
          }
        }
        //Classify by means of a1,a2,a3,a4
        if(a1>=a2 && a2>=a3 && a3>=a4){
            kout = 10+k;
        }
        if(a1>=a2 && a2>=a4 && a4 >=a3){
            kout = 20+k;
        }
        if(a1>=a4 && a4>=a2 && a2>=a3){
            kout = 30+k;
        }
    }//k
    return kout;
}

__global__ void DomainBlockClassify(cuda::PtrStep<uchar> downImage,cuda::PtrStep<uchar> Result){
    int x= blockIdx.x;
    int y= threadIdx.x;
    __shared__ int tmpDown[Rb][N2];
    if(y<Dnum){
        for(int i=0;i<Rb;i++){
            tmpDown[i][y] = downImage(x+i,y);
        }
    }else{
        for(int i=0;i<Rb;i++){
            for(int j=0;j<Rb;j++){
                tmpDown[i][y+j] = downImage(x+i,y+j);
            }
        }
    }
    __syncthreads();
    if(x<Dnum && y<Dnum){
        Result(x,y) = Classify(&tmpDown[0][y],N2);
    }
}

__device__ void calSM(int *sourceR,int* sourceD,float* desS,float* desM,float* desErr){
    int Ud = 32;
    int m = 0;
    int i,j,ks;
    float s;
    float sup = 0.0;
    float sdown = 0.0;
    int tmpR,tmpD;
    float tmperr;
    float err = 0.005;

    //Calculate s,m,k
    for(i=0;i<Rb;i++){
        for(j=0;j<Rb;j++){
            Ud += *(sourceD+i*Rb+j);
            m += *(sourceR+i*Rb+j);
        }
    }
    Ud = Ud/(Rb*Rb);
    m = m/(Rb*Rb);
    for(i=0;i<Rb;i++){
        for(j=0;j<Rb;j++){
            tmpR = *(sourceR+i*Rb+j);
            tmpD = *(sourceD+i*Rb+j);
            sup += (tmpD-Ud)*(tmpR);
            sdown += (tmpD-Ud)*(tmpD-Ud);
        }
    }
    s= ( fabs(sdown)<0.01? 0.0 : sup/sdown);
    ks=(s<-1? 0: s>=2.1?31:(short int)(10.5+s*10));
    s=0.1*ks-1;
    for(i=0;i<Rb;i++){
        for(j=0;j<Rb;j++){
            tmpR = *(sourceR+i*Rb+j);
            tmpD = *(sourceD+i*Rb+j);
            tmperr = s*(tmpD-Ud)+ m - tmpR;
            err += (tmperr*tmperr);
        }
    }
    *desS = (float)ks;
    *desM = (float)m;
    *desErr = err;
}

__device__ float calK(int Rk,int Dk){
    if(Rk==Dk){
        return 0;
    }else if(Rk < 4 && Dk < 4){
        if(Rk<Dk){
            if(Dk-Rk ==1){
                return 1;
            }else if(Dk-Rk ==2){
                return 2;
            }else{
                return 3;
            }
        }else{
            if(Rk-Dk==1){
                return 3;
            }else if(Rk-Dk==2){
                return 2;
            }else{
                return 1;
            }
        }
    }else if(Rk >= 4 && Dk >= 4){
        if(Rk<Dk){
            if(Dk-Rk ==1){
                return 3;
            }else if(Dk-Rk ==2){
                return 2;
            }else{
                return 1;
            }
        }else{
            if(Rk-Dk==1){
                return 1;
            }else if(Rk-Dk==2){
                return 2;
            }else{
                return 3;
            }
        }
    }else if(Rk < 4 && Dk >= 4){
        if(Dk-Rk==4){
            return 4;
        }else if(Dk-Rk==5 || Dk-Rk == 1){
            return 5;
        }else if(Dk-Rk==6 || Dk-Rk == 2){
            return 6;
        }else{
            return 7;
        }
    }else{
        if(Rk-Dk==4){
            return 4;
        }else if(Rk-Dk==5 || Rk-Dk == 1){
            return 5;
        }else if(Rk-Dk==6 || Rk-Dk == 2){
            return 6;
        }else{
            return 7;
        }
    }
}

__global__ static void RangeParallel(cuda::PtrStep<uchar> image,cuda::PtrStep<uchar> downImage,cuda::PtrStep<uchar> klass,float *Output,int Rx,int Ry){
    __shared__ float tmpOutput[5][Dnum];
    __shared__ int tmpDown[Rb][N2];
    int i,j;
    int tmpR[Rb][Rb];
    int perR[Rb][Rb];
    //int tmpD[Rb][Rb];
    int perD[Rb][Rb];
    float s,m,err;
    float* ds = &s;
    float* dm = &m;
    float* derr = &err;
    int Dclass,Rclass;
    int Dk,Rk;
    int mask=1;
    int offset=1;
    int x = blockIdx.x;
    int y = threadIdx.x;
    //Set shared mem
    if(y<Dnum){
        for(i=0;i<Rb;i++){
            tmpDown[i][y] = downImage(x+i,y);
        }
    }else{
        for(i=0;i<Rb;i++){
            for(j=0;j<Rb;j++){
                tmpDown[i][y+j] = downImage(x+i,y+j);
            }
        }
    }
    __syncthreads();
    //Set Range block
    for(i=0;i<Rb;i++){
        for(j=0;j<Rb;j++){
            tmpR[i][j] = image(Rx+i,Ry+j);
        }
    }
    Dclass = klass(x,y)/10;
    Rclass = Classify(&tmpR[0][0],Rb);
    Dk = klass(x,y)%10;
    tmpOutput[4][y] = 6553500;
    
    if(Dclass == Rclass/10){
        Rk = Rclass%10;
        permutation(&tmpR[0][0],&perR[0][0],Rb,Rb,Rk);
        permutation(&tmpDown[0][y],&perD[0][0],N2,Rb,Dk);
        calSM(&perR[0][0],&perD[0][0],ds,dm,derr);
        tmpOutput[0][y] = y;
        tmpOutput[1][y] = calK(Rk,Dk);
        tmpOutput[2][y] = *ds;
        tmpOutput[3][y] = *dm;
        tmpOutput[4][y] = *derr;
    }
    __syncthreads();

    while(offset < Dnum){
        if((y & mask) == 0 && (y+offset) < Dnum){
            if(tmpOutput[4][y+offset] < tmpOutput[4][y]){
                tmpOutput[0][y] = tmpOutput[0][y+offset];
                tmpOutput[1][y] = tmpOutput[1][y+offset];
                tmpOutput[2][y] = tmpOutput[2][y+offset];
                tmpOutput[3][y] = tmpOutput[3][y+offset];
                tmpOutput[4][y] = tmpOutput[4][y+offset];   
            }
            
        }
        offset += offset;
        mask = offset + mask;
        __syncthreads();
    }

    if(y==0){
        Output[x*5]= tmpOutput[0][y];
        Output[x*5+1]= tmpOutput[1][y];
        Output[x*5+2]= tmpOutput[2][y];
        Output[x*5+3]= tmpOutput[3][y];
        Output[x*5+4]= tmpOutput[4][y];
    }
} 


int main(int argc, char** argv){
    if(!InitCUDA()) return 0;
    printf("CUDA initialized.\n");
    clock_t start, end, totaltime;
    size_t free_mem,total_mem;
    hipEvent_t startEvent,stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    float eventTime;
    
    Mat image,downimage;
    float *output; 
    cuda::GpuMat Gpuimage,Gpudownimage;
    cuda::GpuMat Gpuclass(Dnum,Dnum,CV_8U);
    float *GpuOutput;
    hipMalloc((void**)&GpuOutput,sizeof(float)*5*Dnum);
    output = (float*)malloc(sizeof(float)*5*Dnum);
    image = imread(argv[1],0);
    resize(image,downimage,Size(image.cols/2,image.rows/2),0,0,INTER_LINEAR);
    //Open the file for store encoding data
    fstream outfile;
    outfile.open("1024Outcode",ios::out);
    if(!outfile){
        cout << "Open out file fail!!" << endl;
        return 0;
    }
    
    start = clock();
       
    //Encoding
    int i,j,ll;
    int x,y,k,m,s;
    float Emin;
    Emin=6553600;
    Gpuimage.upload(image);
    Gpudownimage.upload(downimage);
    
    //Classify the domain block into 3 class
    hipEventCreate(&startEvent);
    DomainBlockClassify<<<Dnum,Dnum>>>(Gpudownimage,Gpuclass);
    hipEventCreate(&stopEvent);
    hipEventElapsedTime(&eventTime,startEvent,stopEvent);
    cout << "Classify Time : " << eventTime << endl;
    //For each Range, calculate s,m value
    for(i=0;i<N;i+=Rb){
        for(j=0;j<N;j+=Rb){
            RangeParallel<<<Dnum,Dnum>>>(Gpuimage,Gpudownimage,Gpuclass,GpuOutput,i,j);
            hipMemcpy2D(output,sizeof(float)*5,GpuOutput,sizeof(float)*5,sizeof(float)*5,Dnum,hipMemcpyDeviceToHost); 
            for(ll=0;ll<Dnum;ll++){
                if(output[ll*5+4] <= Emin){
                    Emin = output[ll*5+4];
                    x = ll;
                    y = output[ll*5];
                    k= output[ll*5+1]; 
                    s= output[ll*5+2]; 
                    m= output[ll*5+3]; 
                }
            }
            Emin = 6553600;
            outfile << (char)x << (char)y << (char)m << (char)((k<<5)+s);  
        }
    }
    
    //Release the memory
    outfile.close();
    Gpuimage.release();
    Gpudownimage.release();
    Gpuclass.release();
    hipFree(GpuOutput);
    free(output);
    end = clock();

    //Print time and the remain memory
    hipError_t cudaErr;
    totaltime = end-start;
    double sec = (double) totaltime / CLOCKS_PER_SEC;
    cout <<"Time:" << sec << endl;
    cudaErr = hipMemGetInfo(&free_mem, &total_mem);
    if(cudaErr != hipSuccess){ 
        printf("%s in %s at line %d\n", hipGetErrorString(cudaErr), __FILE__, __LINE__); 
    }
    cout << "free:" << free_mem << endl;
    cout << "total:" << total_mem << endl;
    return 0;
}